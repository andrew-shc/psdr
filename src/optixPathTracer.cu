#include "hip/hip_runtime.h"
/*

 * SPDX-FileCopyrightText: Copyright (c) 2019 - 2024  NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <optix.h>

#include "optixPathTracer.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

// for intellisense
#include <optix_device.h>

extern "C"
{
    __constant__ Params params;
}

//------------------------------------------------------------------------------
//
// Orthonormal basis helper
//
//------------------------------------------------------------------------------

struct Onb
{
    __forceinline__ __device__ Onb(const float3 &normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3 &p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

//------------------------------------------------------------------------------
//
// Utility functions
//
//------------------------------------------------------------------------------

static __forceinline__ __device__ RadiancePRD loadClosesthitRadiancePRD()
{
    RadiancePRD prd = {};

    prd.attenuation.x = __uint_as_float(optixGetPayload_0());
    prd.attenuation.y = __uint_as_float(optixGetPayload_1());
    prd.attenuation.z = __uint_as_float(optixGetPayload_2());
    prd.seed = optixGetPayload_3();
    prd.depth = optixGetPayload_4();
    // >> prd.pdf_throughput
    // >> prd.pdf_radiance
    // >> prd.gradient_throughput
    // >> prd.gradient_radiance
    // >> prd.num_params_hit
    prd.pdf_throughput = __uint_as_float(optixGetPayload_5());
    prd.pdf_radiance = __uint_as_float(optixGetPayload_6());
    prd.gradient_throughput.x = __uint_as_float(optixGetPayload_7());
    prd.gradient_throughput.y = __uint_as_float(optixGetPayload_8());
    prd.gradient_throughput.z = __uint_as_float(optixGetPayload_9());
    prd.gradient_radiance.x = __uint_as_float(optixGetPayload_10());
    prd.gradient_radiance.y = __uint_as_float(optixGetPayload_11());
    prd.gradient_radiance.z = __uint_as_float(optixGetPayload_12());
    prd.num_params_hit = optixGetPayload_13();

    return prd;
}

static __forceinline__ __device__ RadiancePRD loadMissRadiancePRD()
{
    RadiancePRD prd = {};
    return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD(RadiancePRD prd)
{
    optixSetPayload_0(__float_as_uint(prd.attenuation.x));
    optixSetPayload_1(__float_as_uint(prd.attenuation.y));
    optixSetPayload_2(__float_as_uint(prd.attenuation.z));

    optixSetPayload_3(prd.seed);
    optixSetPayload_4(prd.depth);

    // >> prd.pdf_throughput
    // >> prd.pdf_radiance
    // >> prd.gradient_throughput
    // >> prd.gradient_radiance
    // >> prd.num_params_hit
    optixSetPayload_5(__float_as_uint(prd.pdf_throughput));
    optixSetPayload_6(__float_as_uint(prd.pdf_radiance));
    optixSetPayload_7(__float_as_uint(prd.gradient_throughput.x));
    optixSetPayload_8(__float_as_uint(prd.gradient_throughput.y));
    optixSetPayload_9(__float_as_uint(prd.gradient_throughput.z));
    optixSetPayload_10(__float_as_uint(prd.gradient_radiance.x));
    optixSetPayload_11(__float_as_uint(prd.gradient_radiance.y));
    optixSetPayload_12(__float_as_uint(prd.gradient_radiance.z));
    optixSetPayload_13(prd.num_params_hit);

    optixSetPayload_14(__float_as_uint(prd.emitted.x));
    optixSetPayload_15(__float_as_uint(prd.emitted.y));
    optixSetPayload_16(__float_as_uint(prd.emitted.z));

    optixSetPayload_17(__float_as_uint(prd.radiance.x));
    optixSetPayload_18(__float_as_uint(prd.radiance.y));
    optixSetPayload_19(__float_as_uint(prd.radiance.z));

    optixSetPayload_20(__float_as_uint(prd.origin.x));
    optixSetPayload_21(__float_as_uint(prd.origin.y));
    optixSetPayload_22(__float_as_uint(prd.origin.z));

    optixSetPayload_23(__float_as_uint(prd.direction.x));
    optixSetPayload_24(__float_as_uint(prd.direction.y));
    optixSetPayload_25(__float_as_uint(prd.direction.z));

    optixSetPayload_26(prd.done);
}

static __forceinline__ __device__ void storeMissRadiancePRD(RadiancePRD prd)
{
    optixSetPayload_14(__float_as_uint(prd.emitted.x));
    optixSetPayload_15(__float_as_uint(prd.emitted.y));
    optixSetPayload_16(__float_as_uint(prd.emitted.z));

    optixSetPayload_17(__float_as_uint(prd.radiance.x));
    optixSetPayload_18(__float_as_uint(prd.radiance.y));
    optixSetPayload_19(__float_as_uint(prd.radiance.z));

    optixSetPayload_26(prd.done);
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3 &p)
{
    // Uniformly sample disk.
    const float r = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}

static __forceinline__ __device__ void basic_sample_hemisphere(const float u1, const float u2, float3 &p)
{
    const float z = u1; // uniform in [0,1]
    const float r = sqrtf(fmaxf(0.0f, 1.0f - z * z));
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);
    p.z = z;
}

static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3 ray_origin,
    float3 ray_direction,
    float tmin,
    float tmax,
    RadiancePRD &prd)
{
    unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18, u19, u20, u21, u22, u23, u24, u25, u26;

    u0 = __float_as_uint(prd.attenuation.x);
    u1 = __float_as_uint(prd.attenuation.y);
    u2 = __float_as_uint(prd.attenuation.z);
    u3 = prd.seed;
    u4 = prd.depth;
    u5 = __float_as_uint(prd.pdf_throughput);
    u6 = __float_as_uint(prd.pdf_radiance);
    u7 = __float_as_uint(prd.gradient_throughput.x);
    u8 = __float_as_uint(prd.gradient_throughput.y);
    u9 = __float_as_uint(prd.gradient_throughput.z);
    u10 = __float_as_uint(prd.gradient_radiance.x);
    u11 = __float_as_uint(prd.gradient_radiance.y);
    u12 = __float_as_uint(prd.gradient_radiance.z);
    u13 = prd.num_params_hit;

    // Note:
    // This demonstrates the usage of the OptiX shader execution reordering
    // (SER) API.  In the case of this computationally simple shading code,
    // there is no real performance benefit.  However, with more complex shaders
    // the potential performance gains offered by reordering are significant.
    optixTraverse(
        PAYLOAD_TYPE_RADIANCE,
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f, // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        0,              // SBT offset
        RAY_TYPE_COUNT, // SBT stride
        0,              // missSBTIndex
        u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18, u19, u20, u21, u22, u23, u24, u25, u26);
    optixReorder(
        // Application specific coherence hints could be passed in here
    );

    optixInvoke(
        PAYLOAD_TYPE_RADIANCE,
        u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18, u19, u20, u21, u22, u23, u24, u25, u26);

    prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
    prd.seed = u3;
    prd.depth = u4;
    prd.pdf_throughput = __uint_as_float(u5);
    prd.pdf_radiance = __uint_as_float(u6);
    prd.gradient_throughput = make_float3(__uint_as_float(u7), __uint_as_float(u8), __uint_as_float(u9));
    prd.gradient_radiance = make_float3(__uint_as_float(u10), __uint_as_float(u11), __uint_as_float(u12));
    prd.num_params_hit = u13;

    prd.emitted = make_float3(__uint_as_float(u14), __uint_as_float(u15), __uint_as_float(u16));
    prd.radiance = make_float3(__uint_as_float(u17), __uint_as_float(u18), __uint_as_float(u19));
    prd.origin = make_float3(__uint_as_float(u20), __uint_as_float(u21), __uint_as_float(u22));
    prd.direction = make_float3(__uint_as_float(u23), __uint_as_float(u24), __uint_as_float(u25));
    prd.done = u26;
}

// Returns true if ray is occluded, else false
static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3 ray_origin,
    float3 ray_direction,
    float tmin,
    float tmax)
{
    // We are only casting probe rays so no shader invocation is needed
    optixTraverse(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax, 0.0f, // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,              // SBT offset
        RAY_TYPE_COUNT, // SBT stride
        0               // missSBTIndex
    );
    return optixHitObjectIsHit();
}

//------------------------------------------------------------------------------
//
// Programs
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int w = params.width;
    const int h = params.height;
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V;
    const float3 W = params.W;
    const float3 W_normalized = normalize(W); // Normalized camera forward direction
    const uint3 idx = optixGetLaunchIndex();
    const int launch_seed = params.launch_seed;

    unsigned int seed = tea<4>(idx.y * w + idx.x, launch_seed);

    float3 result = make_float3(0.0f);
    float3 result_grads = make_float3(0.0f);
    int i = params.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        const float2 d = 2.0f * make_float2(
                                    (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
                                    (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)) -
                         1.0f;
        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;

        // Cosine term between ray direction and camera forward direction
        float cos_vignette = dot(ray_direction, W_normalized);

        RadiancePRD prd;
        prd.attenuation = make_float3(1.f) * cos_vignette;
        prd.seed = seed;
        prd.depth = 0;
        prd.pdf_throughput = 1.0f;
        prd.pdf_radiance = 1.0f;
        prd.gradient_throughput = make_float3(1.f) * cos_vignette;
        prd.gradient_radiance = make_float3(1.f);
        prd.num_params_hit = 0;

        for (;;)
        {
            traceRadiance(
                params.handle,
                ray_origin,
                ray_direction,
                0.01f, // tmin       // TODO: smarter offset
                1e16f, // tmax
                prd);

            result += prd.emitted;
            result += prd.radiance * prd.attenuation / (prd.pdf_radiance * prd.pdf_throughput);

            float r = 0.0f;
            float g = 0.0f;
            float b = 0.0f;
            if (prd.num_params_hit > 1)
            {
                r = powf(params.parameter.x, prd.num_params_hit - 1);
                g = powf(params.parameter.y, prd.num_params_hit - 1);
                b = powf(params.parameter.z, prd.num_params_hit - 1);
            }
            else if (prd.num_params_hit == 1)
            {
                r = 1.0f;
                g = 1.0f;
                b = 1.0f;
            }
            result_grads += (prd.gradient_throughput * prd.gradient_radiance) / (prd.pdf_radiance * prd.pdf_throughput) * prd.num_params_hit * make_float3(r, g, b);

            // Russian roulette using attenuation magnitude as survival probability
            const float survival_prob = length(prd.attenuation) * 0.9;
            const bool russian_roulette_terminate = rnd(prd.seed) > survival_prob;
            const bool done = prd.done || russian_roulette_terminate;

            if (done)
                break;

            // Compensate for Russian roulette
            prd.pdf_throughput *= survival_prob;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ++prd.depth;
        }
    } while (--i);

    const uint3 launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;
    float3 accum_color = result / static_cast<float>(params.samples_per_launch);

    float3 accum_gradients = result_grads / static_cast<float>(params.samples_per_launch);

    // subframe index is used when you're sampling again and again to get better result over time
    // different to what we want

    // if (subframe_index > 0)
    // {
    //     const float a = 1.0f / static_cast<float>(subframe_index + 1);
    //     const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
    //     accum_color = lerp(accum_color_prev, accum_color, a);
    // }
    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_index] = make_color(accum_color);
    params.gradient_buffer[image_index] = make_color(accum_gradients);

    params.frame_buffer_radiance[image_index] = make_float4(accum_color, 1.0f);
    params.gradient_buffer_radiance[image_index] = make_float4(accum_gradients, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

    MissData *rt_data = reinterpret_cast<MissData *>(optixGetSbtDataPointer());
    RadiancePRD prd = loadMissRadiancePRD();

    prd.radiance = make_float3(rt_data->bg_color); // keep 0; not sure if PDFs are needed for this
    prd.emitted = make_float3(0.f);
    prd.done = true;

    storeMissRadiancePRD(prd);
}

extern "C" __global__ void __closesthit__radiance()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

    HitGroupData *rt_data = (HitGroupData *)optixGetSbtDataPointer();

    const int prim_idx = optixGetPrimitiveIndex();
    const float3 ray_dir = optixGetWorldRayDirection();
    const int vert_idx_offset = prim_idx * 3;

    const float3 v0 = make_float3(rt_data->vertices[vert_idx_offset + 0]);
    const float3 v1 = make_float3(rt_data->vertices[vert_idx_offset + 1]);
    const float3 v2 = make_float3(rt_data->vertices[vert_idx_offset + 2]);
    const bool is_parameter = rt_data->is_parameter;
    const float3 N_0 = normalize(cross(v1 - v0, v2 - v0));

    const float3 N = faceforward(N_0, -ray_dir, N_0);
    const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;

    RadiancePRD prd = loadClosesthitRadiancePRD();

    if (is_parameter)
    {
        prd.num_params_hit += 1;
    }

    if (prd.depth == 0)
        prd.emitted = rt_data->emission_color;
    else
        prd.emitted = make_float3(0.0f);

    unsigned int seed = prd.seed;
    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        // basic_sample_hemisphere(z1, z2, w_in);
        cosine_sample_hemisphere(z1, z2, w_in);
        Onb onb(N);
        onb.inverse_transform(w_in);
        prd.direction = w_in;
        prd.origin = P;

        // float cos_theta = dot(w_in, N);
        // float3 f = rt_data->diffuse_color / M_PIf * cos_theta;
        float3 f = rt_data->diffuse_color;
        prd.attenuation *= f;
        prd.pdf_throughput *= 1.0; // cosine-weighted sampling PDF cancels with BRDF*cos

        if (!is_parameter)
            prd.gradient_throughput *= f;
    }

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    ParallelogramLight light = params.light;

    // light sampling only when the final depth is reached OR light is accessible

    const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float Ldist = length(light_pos - P);
    const float3 L = normalize(light_pos - P);
    const float nDl = dot(N, L);
    const float LnDl = -dot(light.normal, L);

    const float A = length(cross(light.v1, light.v2));
    const bool occluded =
        traceOcclusion(
            params.handle,
            P,
            L,
            0.01f,          // tmin
            Ldist - 0.01f); // tmax

    //           MAX DEPTH
    // if (prd.depth >= 20 || !occluded)

    prd.radiance = make_float3(0.0f);
    prd.gradient_radiance = make_float3(0.0f);
    prd.pdf_radiance = 1.0f;
    if (!occluded && (nDl > 0.0f && LnDl > 0.0f))
    {

        float G = nDl * LnDl / (M_PIf * Ldist * Ldist);

        float3 LeG = light.emission * G;
        prd.radiance = LeG;
        prd.gradient_radiance = LeG; // if (!is_parameter) // assume the light is never the parameter (for now)
        prd.pdf_radiance *= 1.0f / A;
    }

    prd.done = false;

    storeClosesthitRadiancePRD(prd);
}
